
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>

const unsigned long long THREADS = 32;
const unsigned long long BLOCK_SIZE = 256;

__global__ void pi_MC(float *totals, const int iterations);
int main(int argc, char **argv)
{
    if (argc != 2)
    {
        printf("Usage: ./PiMC <iterations>\n");
        return 1;
    }
    int iterations = atoi(argv[1]);
    if (iterations <= 0)
    {
        printf("Error: Number of iterations must be a positive integer.\n");
        return 1;
    }

    float *dev_a, *dev_b;
    dev_a = (float *)malloc(sizeof(float) * BLOCK_SIZE);
    hipMalloc(&dev_b, sizeof(float) * BLOCK_SIZE);
    // Create CUDA events to measure the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Call the Pi Kernel
    hipEventRecord(start);

    pi_MC<<<BLOCK_SIZE, THREADS>>>(dev_b, iterations);
    hipEventRecord(stop);

    hipMemcpy(dev_a, dev_b, sizeof(float) * BLOCK_SIZE, hipMemcpyDeviceToHost);
    float elapsedTime;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipFree(dev_b);
    printf("Time taken by program: %f seconds\n", elapsedTime / 1000);
    FILE *fp = fopen("cu_exec_time.csv", "a");
    if (fp == NULL)
    {
        printf("Error opening file\n");
        return 1;
    }

    fprintf(fp, "%.8f,%d\n", elapsedTime / 1000, iterations);
    fclose(fp);

    float count = 0;
    for (int i = 0; i < BLOCK_SIZE; i++)
    {
        count += dev_a[i];
    }
    unsigned long long tests = BLOCK_SIZE * iterations * THREADS;

    printf("Estimate of pi using CUDA: %.8f\n", 4.0 * count / tests);

    free(dev_a);

    return 0;
}
__global__ void pi_MC(float *totals, const int iterations)
{
    __shared__ float count[THREADS];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState_t states;
    hiprand_init(clock64(), tid, 0, &states);
    count[threadIdx.x] = 0;
    for (int i = 0; i < iterations; i++)
    {
        float x = hiprand_uniform(&states);
        float y = hiprand_uniform(&states);
        count[threadIdx.x] += 1 - int(x * x + y * y);
    }
    __syncthreads();
    for (int j = 1; j < blockDim.x; j *= 2)
    {
        int idx = 2 * j * threadIdx.x;
        if (idx < blockDim.x)
        {
            count[idx] += count[idx + j];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0)
    {
        atomicAdd(&totals[blockIdx.x], count[0]);
    }
}